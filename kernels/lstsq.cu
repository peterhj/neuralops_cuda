
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

__global__ void ind_lst_sq_fwd_kernel(
    const float *x,
    uint32_t dim,
    uint32_t batch_sz,
    const float *targets,
    const uint32_t *labels,
    const float *weights,
    float *loss)
{
  uint32_t batch_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (batch_idx < batch_sz) {
    uint32_t label_k = labels[batch_idx];
    if (label_k < dim) {
      uint32_t idx = label_k + dim * batch_idx;
      float dx = x[idx] - targets[batch_idx];
      //loss[batch_idx] = 0.5f * weights[batch_idx] * dx * dx;
      loss[batch_idx] = 0.5f * dx * dx;
    } else {
      loss[batch_idx] = 0.0f;
    }
  }
}

extern "C" void neuralops_cuda_ind_lst_sq_fwd(
    const float *x,
    size_t dim,
    size_t batch_sz,
    const float *targets,
    const uint32_t *labels,
    const float *weights,
    float *loss,
    hipStream_t stream)
{
  ind_lst_sq_fwd_kernel<<<(batch_sz+1024-1)/1024, 1024, 0, stream>>>(
      x, dim, batch_sz, targets, labels, weights, loss);
}

__global__ void ind_lst_sq_bwd_kernel(
    const float *x,
    uint32_t dim,
    uint32_t batch_sz,
    const float *targets,
    const uint32_t *labels,
    const float *weights,
    float *grad)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t k = idx % dim;
  uint32_t batch_idx = idx / dim;
  if (k < dim && batch_idx < batch_sz) {
    if (k == labels[batch_idx]) {
      grad[idx] = x[idx] - targets[batch_idx];
    } else {
      grad[idx] = 0.0f;
    }
  }
}

extern "C" void neuralops_cuda_ind_lst_sq_bwd(
    const float *x,
    size_t dim,
    size_t batch_sz,
    const float *targets,
    const uint32_t *labels,
    const float *weights,
    float *grad,
    hipStream_t stream)
{
  uint32_t n = dim * batch_sz;
  ind_lst_sq_bwd_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      x, dim, batch_sz, targets, labels, weights, grad);
}
