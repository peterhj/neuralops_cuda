
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

__global__ void activate_rect_fwd_kernel(
    const float *in_act,
    uint32_t dim,
    float *out_act)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_act[idx];
    out_act[idx] = x * (x > 0.0f);
  }
}

extern "C" void neuralops_cuda_activate_rect_fwd(
    const float *in_act,
    size_t dim,
    float *out_act,
    hipStream_t stream)
{
  activate_rect_fwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_act, dim, out_act);
}

__global__ void activate_rect_bwd_kernel(
    const float *in_act,
    uint32_t dim,
    const float *out_delta,
    float *in_delta)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_act[idx];
    in_delta[idx] = out_delta[idx] * (x > 0.0f);
  }
}

extern "C" void neuralops_cuda_activate_rect_bwd(
    const float *in_act,
    size_t dim,
    const float *out_delta,
    float *in_delta,
    hipStream_t stream)
{
  activate_rect_bwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_act, dim, out_delta, in_delta);
}

__global__ void activate_rect_bwd2_kernel(
    const float *in_act,
    uint32_t dim,
    const float *out_delta2,
    float *in_delta2)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_act[idx];
    in_delta2[idx] = out_delta2[idx] * (x > 0.0f);
  }
}

extern "C" void neuralops_cuda_activate_rect_bwd2(
    const float *in_act,
    size_t dim,
    const float *out_delta2,
    float *in_delta2,
    hipStream_t stream)
{
  activate_rect_bwd2_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_act, dim, out_delta2, in_delta2);
}

__global__ void activate_rect_rfwd_kernel(
    const float *in_val,
    uint32_t dim,
    const float *in_r_val,
    float *out_r_val)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_val[idx];
    out_r_val[idx] = in_r_val[idx] * (x > 0.0f);
  }
}

extern "C" void neuralops_cuda_activate_rect_rfwd(
    const float *in_val,
    size_t dim,
    const float *in_r_val,
    float *out_r_val,
    hipStream_t stream)
{
  activate_rect_rfwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_val, dim, in_r_val, out_r_val);
}

__global__ void activate_rect_rbwd_kernel(
    const float *in_val,
    uint32_t dim,
    const float *out_r_grad,
    float *in_r_grad)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_val[idx];
    in_r_grad[idx] = out_r_grad[idx] * (x > 0.0f);
  }
}

extern "C" void neuralops_cuda_activate_rect_rbwd(
    const float *in_val,
    size_t dim,
    const float *out_r_grad,
    float *in_r_grad,
    hipStream_t stream)
{
  activate_rect_rbwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_val, dim, out_r_grad, in_r_grad);
}

__global__ void activate_leakrect_fwd_kernel(
    const float *in_act,
    uint32_t dim,
    float *out_act,
    float neg_slope)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_act[idx];
    int mask = x > 0.0f;
    out_act[idx] = x * (neg_slope * (1 - mask) + mask);
  }
}

extern "C" void neuralops_cuda_activate_leakrect_fwd(
    const float *in_act,
    size_t dim,
    float *out_act,
    float neg_slope,
    hipStream_t stream)
{
  activate_leakrect_fwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_act, dim, out_act, neg_slope);
}

__global__ void activate_leakrect_bwd_kernel(
    const float *in_act,
    uint32_t dim,
    const float *out_delta,
    float *in_delta,
    float neg_slope)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_act[idx];
    int mask = x > 0.0f;
    float dy = out_delta[idx];
    in_delta[idx] = dy * (neg_slope * (1 - mask) + mask);
  }
}

extern "C" void neuralops_cuda_activate_leakrect_bwd(
    const float *in_act,
    size_t dim,
    const float *out_delta,
    float *in_delta,
    float neg_slope,
    hipStream_t stream)
{
  activate_leakrect_bwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_act, dim, out_delta, in_delta, neg_slope);
}

__global__ void activate_logistic_fwd_kernel(
    const float *in_buf,
    uint32_t dim,
    float *out_buf)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_buf[idx];
    out_buf[idx] = 1.0f / (1.0f + expf(-x));
  }
}

extern "C" void neuralops_cuda_activate_logistic_fwd(
    const float *in_buf,
    size_t dim,
    float *out_buf,
    hipStream_t stream)
{
  activate_logistic_fwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_buf, dim, out_buf);
}

__global__ void activate_logistic_bwd_kernel(
    const float *in_buf,
    uint32_t dim,
    const float *out_delta,
    float *in_delta)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < dim) {
    float x = in_buf[idx];
    float y = 1.0f / (1.0f + expf(-x));
    in_delta[idx] = y * (1.0f - y) * out_delta[idx];
  }
}

extern "C" void neuralops_cuda_activate_logistic_bwd(
    const float *in_buf,
    size_t dim,
    const float *out_delta,
    float *in_delta,
    hipStream_t stream)
{
  activate_logistic_bwd_kernel<<<(dim+1024-1)/1024, 1024, 0, stream>>>(
      in_buf, dim, out_delta, in_delta);
}
