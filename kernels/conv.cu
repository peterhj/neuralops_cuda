
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

#define OFFSET_BANK(idx) ({ __typeof__ (idx) _idx = idx; ((_idx) + ((_idx) / 32)); })

__global__ void im2col_gpu_kernel(
    const int n,
    const float* data_im,
    const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    float* data_col)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    float* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const float* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr =
            (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
            data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

extern "C" void neuralops_cuda_caffe_im2col(
    const float* data_im,
    const int channels,
    const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    float* data_col,
    hipStream_t stream)
{
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h -
      (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w -
      (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<<<(num_kernels+1024-1)/1024, 1024, 0, stream>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
      width_col, data_col);
}

__global__ void col2im_gpu_kernel(
    const int n,
    const float* data_col,
    const int height, const int width, const int channels,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    float* data_im)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < n) {
    float val = 0;
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
                                height_col + h_col) * width_col + w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

extern "C" void neuralops_cuda_caffe_col2im(
    const float* data_col,
    const int channels,
    const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    float* data_im,
    hipStream_t stream)
{
  int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
      stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
      stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<<<(num_kernels+1024-1)/1024, 1024, 0, stream>>>(
      num_kernels, data_col, height, width, channels, kernel_h, kernel_w,
      pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
      height_col, width_col, data_im);
}

__global__ void conv_diag_affine_fwd_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    const float *bias,
    float *out_act)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int u = idx % spatial_dim;
  int c = (idx / spatial_dim) % num_channels;
  int batch_idx = idx / (spatial_dim * num_channels);
  if (u < spatial_dim && c < num_channels && batch_idx < batch_size) {
    float gamma = scale[c];
    float beta = bias[c];
    float y = gamma * in_act[idx] + beta;
    out_act[idx] = y;
  }
}

extern "C" void neuralops_cuda_conv2d_scale_fwd(
    const float *in_act,
    size_t spatial_dim,
    size_t num_channels,
    size_t batch_size,
    const float *scale,
    const float *bias,
    float *out_act,
    hipStream_t stream)
{
  int n = spatial_dim * num_channels * batch_size;
  conv_diag_affine_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_act, spatial_dim, num_channels, batch_size, scale, bias, out_act);
}

__global__ void conv_diag_affine_bwd_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *out_delta,
    const float *scale,
    float *scale_grad,
    float *bias_grad,
    float *in_delta)
{
  __shared__ float scale_grad_cache[1024+32];
  __shared__ float bias_grad_cache[1024+32];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int bank_idx = OFFSET_BANK(threadIdx.x);
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int warp_idx = idx % 32;
  int c = (idx / 32) % num_channels;
  int u0 = warp_idx + ((idx / (32 * num_channels)) % block_spatial_dim) * (16*32);
  int batch_idx = idx / (32 * num_channels * block_spatial_dim);
  if (c < num_channels && u0 < spatial_dim && batch_idx < batch_size) {
    float gamma = scale[c];
    float d_gamma = 0.0f;
    float d_beta = 0.0f;
    int i0 = c * spatial_dim + batch_idx * spatial_dim * num_channels;
    int u_limit = min(spatial_dim, u0 + 16*32);
    for (int u = u0; u < u_limit; u += 32) {
      int i = i0 + u;
      float dy = out_delta[i];
      d_gamma += dy * in_act[i];
      d_beta += dy;
      in_delta[i] = dy * gamma;
      //in_delta[i] += dy * gamma;
      //atomicAdd(&in_delta[i], dy * gamma);
    }
    scale_grad_cache[bank_idx] = d_gamma;
    bias_grad_cache[bank_idx] = d_beta;
  } else {
    scale_grad_cache[bank_idx] = 0.0f;
    bias_grad_cache[bank_idx] = 0.0f;
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (threadIdx.x % 2 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+1];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+1];
    }
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (threadIdx.x % 4 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+2];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+2];
    }
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (threadIdx.x % 8 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+4];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+4];
    }
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (threadIdx.x % 16 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+8];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+8];
    }
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (threadIdx.x % 32 == 0 && u0 < spatial_dim) {
      float d_gamma = scale_grad_cache[bank_idx] + scale_grad_cache[bank_idx+16];
      atomicAdd(&scale_grad[c], d_gamma);
      float d_beta = bias_grad_cache[bank_idx] + bias_grad_cache[bank_idx+16];
      atomicAdd(&bias_grad[c], d_beta);
    }
  }
}

extern "C" void neuralops_cuda_conv2d_scale_bwd(
    const float *in_act,
    size_t spatial_dim,
    size_t num_channels,
    size_t batch_size,
    const float *out_delta,
    const float *scale,
    float *scale_grad,
    float *bias_grad,
    float *in_delta,
    hipStream_t stream)
{
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int n = 32 * num_channels * block_spatial_dim * batch_size;
  conv_diag_affine_bwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_act, spatial_dim, num_channels, batch_size, out_delta, scale, scale_grad, bias_grad, in_delta);
}

__global__ void conv_scale_rfwd_kernel(
    const float *in_val,
    uint32_t spatial_dim,
    uint32_t num_channels,
    uint32_t batch_size,
    const float *in_r_val,
    const float *scale,
    const float *scale_r_dir,
    const float *bias_r_dir,
    float *out_r_val)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t u = idx % spatial_dim;
  uint32_t c = (idx / spatial_dim) % num_channels;
  uint32_t batch_idx = idx / (spatial_dim * num_channels);
  if (u < spatial_dim && c < num_channels && batch_idx < batch_size) {
    float alpha = scale[c];
    float r_alpha = scale_r_dir[c];
    float r_beta = bias_r_dir[c];
    float r_y = alpha * in_r_val[idx] + r_alpha * in_val[idx] + r_beta;
    out_r_val[idx] = r_y;
  }
}

extern "C" void neuralops_cuda_conv_scale_rfwd(
    const float *in_val,
    size_t spatial_dim,
    size_t num_channels,
    size_t batch_size,
    const float *in_r_val,
    const float *scale,
    const float *scale_r_dir,
    const float *bias_r_dir,
    float *out_r_val,
    hipStream_t stream)
{
  uint32_t n = spatial_dim * num_channels * batch_size;
  conv_scale_rfwd_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_val, spatial_dim, num_channels, batch_size, in_r_val, scale, scale_r_dir, bias_r_dir, out_r_val);
}
